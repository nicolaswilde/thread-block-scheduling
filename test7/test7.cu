#include <kernels.h>
#include <archs.h>

int main() {

    const int KN = 1;

    clock_t clock_rate = CLOCK;

    int b[KN], t[KN], r[KN], s[KN];

    b[0] = N_SM;
    t[0] = 32;
    r[0] = 32;
    s[0] = 0;

    printf("================================================================================\n");
    printf("Test7 ==========================================================================\n");
    printf("K1 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream1\n", b[0], t[0], r[0], s[0]);
    printf("================================================================================\n");

    // smid

    size_t smid_size[KN];
    for (int i = 0; i < KN; i++) smid_size[i] = b[i] * sizeof(int);

    int *d_smid[KN], *h_smid[KN];
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMalloc(&d_smid[i], smid_size[i]));
        h_smid[i] = (int *)malloc(smid_size[i]);
        checkCuda(hipMemset(d_smid[i], 0xffffffff, smid_size[i]));
    }

    // seconds

    size_t seconds_size[KN];
    for (int i = 0; i < KN; i++) seconds_size[i] = b[i] * sizeof(float);

    float *d_seconds[KN], *h_seconds[KN];
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMalloc(&d_seconds[i], seconds_size[i]));
        h_seconds[i] = (float *)malloc(seconds_size[i]);
        for (int j = 0; j < b[i]; j++) h_seconds[i][j] = 1.0f;
        checkCuda(hipMemcpy(d_seconds[i], h_seconds[i], seconds_size[i], hipMemcpyHostToDevice));
    }

    // stream

    hipStream_t stream[KN];
    for (int i = 0; i < KN; i++) checkCuda(hipStreamCreate(&stream[i]));

    // invoke kernels

    for (int i = 0; i < KN; i++)
        invoke_kernel(b[i], t[i], r[i], s[i], stream[i], d_seconds[i], clock_rate, d_smid[i]);

    // smid

    for (int i = 0; i < KN; i++) {
        checkCuda(hipMemcpy(h_smid[i], d_smid[i], smid_size[i], hipMemcpyDeviceToHost));

        printf("K%d smid = %d", i + 1, h_smid[i][0]);
        for (int j = 1; j < b[i]; j++) printf(", %d", h_smid[i][j]);
        printf("\n");
    }

    printf("================================================================================\n");

    // free

    for (int i = 0; i < KN; i++) {
        checkCuda(hipFree(d_smid[i]));
        checkCuda(hipFree(d_seconds[i]));
        free(h_smid[i]);
        free(h_seconds[i]);
        checkCuda(hipStreamDestroy(stream[i]));
    }
}
