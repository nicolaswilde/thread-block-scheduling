#include <kernels.h>
#include <archs.h>

int main() {

    const int KN = 9;

    clock_t clock_rate = CLOCK;

    int b[KN], t[KN], r[KN], s[KN];

    b[0] = N_SM; t[0] = 32; r[0] = 32; s[0] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[1] = N_SM; t[1] = 32; r[1] = 32; s[1] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[2] = N_SM; t[2] = 32; r[2] = 32; s[2] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[3] = N_SM; t[3] = 32; r[3] = 32; s[3] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[4] = N_SM; t[4] = 32; r[4] = 32; s[4] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[5] = N_SM; t[5] = 32; r[5] = 32; s[5] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[6] = N_SM; t[6] = 32; r[6] = 32; s[6] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[7] = N_SM; t[7] = 32; r[7] = 32; s[7] = ((MAX_SMEM_PER_SM / 1024 / 8) - 1) * 1024;
    b[8] = N_SM; t[8] = 32; r[8] = 32; s[8] = ((MAX_SMEM_PER_SM / 1024 / 2) - 1) * 1024;
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_r32), hipFuncAttributeMaxDynamicSharedMemorySize, s[8]);

    printf("================================================================================\n");
    printf("Test14 =========================================================================\n");
    printf("K1 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream1\n", b[0], t[0], r[0], s[0]);
    printf("K2 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream2\n", b[1], t[1], r[1], s[1]);
    printf("K3 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream3\n", b[2], t[2], r[2], s[2]);
    printf("K4 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream4\n", b[3], t[3], r[3], s[3]);
    printf("K5 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream5\n", b[4], t[4], r[4], s[4]);
    printf("K6 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream6\n", b[5], t[5], r[5], s[5]);
    printf("K7 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream7\n", b[6], t[6], r[6], s[6]);
    printf("K8 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream8\n", b[7], t[7], r[7], s[7]);
    printf("K9 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream9\n", b[8], t[8], r[8], s[8]);
    printf("================================================================================\n");

    // smid

    size_t smid_size[KN];
    for (int i = 0; i < KN; i++) smid_size[i] = b[i] * sizeof(int);

    int *d_smid[KN], *h_smid[KN];
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMalloc(&d_smid[i], smid_size[i]));
        h_smid[i] = (int *)malloc(smid_size[i]);
        checkCuda(hipMemset(d_smid[i], 0xffffffff, smid_size[i]));
    }

    // seconds

    size_t seconds_size[KN];
    for (int i = 0; i < KN; i++) seconds_size[i] = b[i] * sizeof(float);

    float *d_seconds[KN], *h_seconds[KN];
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMalloc(&d_seconds[i], seconds_size[i]));
        h_seconds[i] = (float *)malloc(seconds_size[i]);
        for (int j = 0; j < b[i]; j++) {
            if (i % 2 == 0)
                h_seconds[i][j] = 1.0f;
            else
                h_seconds[i][j] = 2.0f;
        }
        checkCuda(hipMemcpy(d_seconds[i], h_seconds[i], seconds_size[i], hipMemcpyHostToDevice));
    }

    // stream

    hipStream_t stream[KN];
    for (int i = 0; i < KN; i++) checkCuda(hipStreamCreate(&stream[i]));

    // invoke kernels

    for (int i = 0; i < KN; i++)
        invoke_kernel(b[i], t[i], r[i], s[i], stream[i], d_seconds[i], clock_rate, d_smid[i]);

    // smid
    /*
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMemcpy(h_smid[i], d_smid[i], smid_size[i], hipMemcpyDeviceToHost));

        printf("K%d smid = %d", i + 1, h_smid[i][0]);
        for (int j = 1; j < b[i]; j++) printf(", %d", h_smid[i][j]);
        printf("\n");
    }

    printf("================================================================================\n");
    */

    // free

    for (int i = 0; i < KN; i++) {
        checkCuda(hipFree(d_smid[i]));
        checkCuda(hipFree(d_seconds[i]));
        free(h_smid[i]);
        free(h_seconds[i]);
        checkCuda(hipStreamDestroy(stream[i]));
    }
}
