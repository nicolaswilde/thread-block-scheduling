#include <kernels.h>
#include <archs.h>

int main() {

    const int KN = 4;

    clock_t clock_rate = CLOCK;

    int b[KN], t[KN], r[KN], s[KN];

    b[0] = N_SM * (MAX_THREAD_PER_SM / MAX_THREAD_PER_THREAD_BLOCK);
    t[0] = MAX_THREAD_PER_THREAD_BLOCK;
    r[0] = 32;
    s[0] = 0;

    b[1] = N_SM * (MAX_THREAD_PER_SM / MAX_THREAD_PER_THREAD_BLOCK);
    t[1] = MAX_THREAD_PER_THREAD_BLOCK;
    r[1] = 32;
    s[1] = 0;

    b[2] = N_SM * (MAX_THREAD_PER_SM / MAX_THREAD_PER_THREAD_BLOCK);
    t[2] = MAX_THREAD_PER_THREAD_BLOCK;
    r[2] = 32;
    s[2] = 0;

    b[3] = N_SM * (MAX_THREAD_PER_SM / MAX_THREAD_PER_THREAD_BLOCK);
    t[3] = MAX_THREAD_PER_THREAD_BLOCK;
    r[3] = 32;
    s[3] = 0;

    printf("================================================================================\n");
    printf("Test4 ==========================================================================\n");
    printf("K1 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream1\n", b[0], t[0], r[0], s[0]);
    printf("K2 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream1\n", b[1], t[1], r[1], s[1]);
    printf("K3 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream2\n", b[2], t[2], r[2], s[2]);
    printf("K4 = K(%d, %d, %d, %d Bytes), all blocks 1s, stream3\n", b[3], t[3], r[3], s[3]);
    printf("================================================================================\n");

    // smid

    size_t smid_size[KN];
    for (int i = 0; i < KN; i++) smid_size[i] = b[i] * sizeof(int);

    int *d_smid[KN], *h_smid[KN];
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMalloc(&d_smid[i], smid_size[i]));
        h_smid[i] = (int *)malloc(smid_size[i]);
        checkCuda(hipMemset(d_smid[i], 0xffffffff, smid_size[i]));
    }

    // seconds

    size_t seconds_size[KN];
    for (int i = 0; i < KN; i++) seconds_size[i] = b[i] * sizeof(float);

    float *d_seconds[KN], *h_seconds[KN];
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMalloc(&d_seconds[i], seconds_size[i]));
        h_seconds[i] = (float *)malloc(seconds_size[i]);
        for (int j = 0; j < b[i]; j++) h_seconds[i][j] = 1.0f;
        checkCuda(hipMemcpy(d_seconds[i], h_seconds[i], seconds_size[i], hipMemcpyHostToDevice));
    }

    // stream

    hipStream_t stream[KN];
    for (int i = 0; i < KN; i++) checkCuda(hipStreamCreate(&stream[i]));

    // invoke kernels

    invoke_kernel(b[0], t[0], r[0], s[0], stream[0], d_seconds[0], clock_rate, d_smid[0]);
    invoke_kernel(b[1], t[1], r[1], s[1], stream[0], d_seconds[1], clock_rate, d_smid[1]);
    invoke_kernel(b[2], t[2], r[2], s[2], stream[1], d_seconds[2], clock_rate, d_smid[2]);
    invoke_kernel(b[3], t[3], r[3], s[3], stream[2], d_seconds[3], clock_rate, d_smid[3]);

    // smid
    /*
    for (int i = 0; i < KN; i++) {
        checkCuda(hipMemcpy(h_smid[i], d_smid[i], smid_size[i], hipMemcpyDeviceToHost));

        printf("K%d smid = %d", i + 1, h_smid[i][0]);
        for (int j = 1; j < b[i]; j++) printf(", %d", h_smid[i][j]);
        printf("\n");
    }

    printf("================================================================================\n");
    */

    // free

    for (int i = 0; i < KN; i++) {
        checkCuda(hipFree(d_smid[i]));
        checkCuda(hipFree(d_seconds[i]));
        free(h_smid[i]);
        free(h_seconds[i]);
        checkCuda(hipStreamDestroy(stream[i]));
    }
}
