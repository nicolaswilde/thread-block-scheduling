#include "hip/hip_runtime.h"
#include <kernels.h>

__global__ void kernel_r184(int *smid, int smem_size, float *seconds, clock_t clock_rate) {

    clock_t t0 = clock64();

    int this_smid;
    asm("mov.u32 %0, %smid;" : "=r"(this_smid));

    if (threadIdx.x == 0) smid[blockIdx.x] = this_smid;

    extern __shared__ int smem[];

    const int regs_count = 256;
    const int smem_count = smem_size / sizeof(int);
    int regs[regs_count];

    #pragma unroll 256
    for (int i = 0; i < smem_count; i++) regs[i % regs_count] += smem[i];
    for (int i = 0; i < regs_count && i < smem_count; i++) smem[i] = regs[i];

    float this_seconds = seconds[blockIdx.x];
    clock_t t1 = clock64();
    while ((t1 - t0) / (clock_rate * 1000.0f) < this_seconds) t1 = clock64();
}
