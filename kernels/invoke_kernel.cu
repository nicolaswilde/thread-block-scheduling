#include <kernels.h>

void invoke_kernel(int b, int t, int r, int s, hipStream_t stream, float *seconds, clock_t clock_rate, int *smid) {
    switch (r) {
        case  24: kernel_r24 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  32: kernel_r32 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  40: kernel_r40 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  41: kernel_r41 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  48: kernel_r48 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  56: kernel_r56 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  64: kernel_r64 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  72: kernel_r72 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  80: kernel_r80 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  88: kernel_r88 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case  96: kernel_r96 <<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 104: kernel_r104<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 112: kernel_r112<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 120: kernel_r120<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 128: kernel_r128<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 136: kernel_r136<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 144: kernel_r144<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 152: kernel_r152<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 160: kernel_r160<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 168: kernel_r168<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 176: kernel_r176<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 184: kernel_r184<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 192: kernel_r192<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 200: kernel_r200<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 208: kernel_r208<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 216: kernel_r216<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 224: kernel_r224<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 232: kernel_r232<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 240: kernel_r240<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 248: kernel_r248<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        case 255: kernel_r255<<<b, t, s, stream>>>(smid, s, seconds, clock_rate); break;
        default: printf("ERROR: invoke kernel r = %d\n", r);
    }
}
